
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorMatrixMult(const float *A, const float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float sum = 0.0f; // Initialize sum to zero
        // Perform the matrix-vector multiplication
        for (int j = 0; j < N; j++)
        {
            sum += A[i * N + j] * B[j]; // [i * N + j] accesses the i-th row of matrix A i * N means the i-th row, j is the column index
            // B[j] accesses the j-th element of vector B 
        }
        C[i] = sum;
    }
}

int main()
{
    // initialize the matrix
    const int N = 10;
    float *A, *B, *C;

    // initialize the input matrices
    A = (float *)malloc(N * N * sizeof(float));
    B = (float *)malloc(N * sizeof(float));
    C = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = 1.0f;
        }
        B[i] = 2.0f;
        C[i] = 0.0f;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMemcpy(d_a, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);
    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;
    vectorMatrixMult<<<gridsize, blocksize>>>(d_a, d_b, d_c, N);

    hipDeviceSynchronize();
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("A:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {

            printf("%.2f ", A[i * N + j]); // Prints each element with 2 decimal precision
        }
        printf("\n"); // Adds a newline after each row
    }

    printf("C:\n");
    for (int i = 0; i < N; i++)
    {

        printf("%.2f ", C[i]); // Prints each element with 2 decimal precision
    }
    printf("\n");
    printf("B:\n");
    for (int i = 0; i < N; i++)
    {

        printf("%.2f ", B[i]); // Prints each element with 2 decimal precision
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}