
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectAdd(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int N = 10;
    float a[N], b[N], c[N];
    float *d_a, *d_b, *d_c;

    // Initialize vectors a and b
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = ceil(N / (float)blockSize);
    vectAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}